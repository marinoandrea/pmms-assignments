#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <errno.h>
#include <getopt.h>
#include "timer.h"

#define filter_height 5
#define filter_width 5

#define border_height ((filter_height/2)*2)
#define border_width ((filter_width/2)*2)

#define block_size_x 32
#define block_size_y 16

#define SEED 1234

using namespace std;

void convolutionSeq(float *output, float *input, float *filter,
                    long int image_height, long int image_width,
                    long int input_height, long int input_width) {
    timer sequentialTime = timer("Sequential");
    
    sequentialTime.start();

    // For each pixel in the output image
    for (int y=0; y < image_height; y++) {
        for (int x=0; x < image_width; x++) { 
            output[y*image_width+x]=0;
            
            // For each filter weight
            for (int i=0; i < filter_height; i++) {
                for (int j=0; j < filter_width; j++) {
                    output[y*image_width+x] += input[(y+i)*input_width+x+j] * filter[i*filter_width+j];
                }
            }

	        output[y*image_width+x] /= 35;
        }
    }
  
    sequentialTime.stop(); 
    cout << "convolution (sequential): \t\t" << sequentialTime << endl;

}

__global__ void convolution_kernel_naive(float *output, float *input, float *filter,
                                         long int image_height, long int image_width,
                                         long int input_height, long int input_width) {
    // TODO: Determine x and y based on Block ID and Grid ID.
    size_t y = blockIdx.y * blockDim.y + threadIdx.y;
    size_t x = blockIdx.x * blockDim.x + threadIdx.x;

    // For each filter weight
    for (size_t i=0; i < filter_height; i++) {
        for (size_t j=0; j < filter_width; j++) {
            output[y*image_width+x] += input[(y+i)*input_width+x+j] * filter[i*filter_width+j];
        }
    }
    
    output[y*image_width+x] /= 35;
}

void convolutionCUDA(float *output, float *input, float *filter,
                    long int image_height, long int image_width,
                    long int input_height, long int input_width) {
    float *d_input; float *d_output; float *d_filter;
    hipError_t err;
    timer kernelTime = timer("kernelTime");
    timer memoryTime = timer("memoryTime");

    // memory allocation
    err = hipMalloc((void **)&d_input, input_height*input_width*sizeof(float));
    if (err != hipSuccess) { fprintf(stderr, "Error in hipMalloc d_input: %s\n", hipGetErrorString( err )); }
    err = hipMalloc((void **)&d_output, image_height*image_width*sizeof(float));
    if (err != hipSuccess) { fprintf(stderr, "Error in hipMalloc d_output: %s\n", hipGetErrorString( err )); }
    err = hipMalloc((void **)&d_filter, filter_height*filter_width*sizeof(float));
    if (err != hipSuccess) { fprintf(stderr, "Error in hipMalloc d_filter: %s\n", hipGetErrorString( err )); }

    memoryTime.start();
    // host to device 
    err = hipMemcpy(d_input, input, input_height*input_width*sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess) { fprintf(stderr, "Error in hipMemcpy host to device input: %s\n", hipGetErrorString( err ));  }
    err = hipMemcpy(d_filter, filter, filter_height*filter_width*sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess) { fprintf(stderr, "Error in hipMemcpy host to device filter: %s\n", hipGetErrorString( err ));  }
    
    // zero the result array 
    err = hipMemset(d_output, 0, image_height*image_width*sizeof(float));
    if (err != hipSuccess) { fprintf(stderr, "Error in hipMemset output: %s\n", hipGetErrorString( err ));  }
    memoryTime.stop();
    //setup the grid and thread blocks
    //thread block size
    dim3 threads(block_size_x, block_size_y);
    //problem size divided by thread block size rounded up
    dim3 grid(int(ceilf(image_width/(float)threads.x)), int(ceilf(image_height/(float)threads.y)) );

    //measure the GPU function
    kernelTime.start();
    convolution_kernel_naive<<<grid, threads>>>(d_output, d_input, d_filter,
                                                image_height, image_width,
                                                input_height, input_width);
    hipDeviceSynchronize();
    kernelTime.stop();
 
    //check to see if all went well
    err = hipGetLastError();
    if (err != hipSuccess) { fprintf(stderr, "Error during kernel launch convolution_kernel: %s\n", hipGetErrorString( err )); }

    //copy the result back to host memory
    memoryTime.start();
    err = hipMemcpy(output, d_output, image_height*image_width*sizeof(float), hipMemcpyDeviceToHost);
    memoryTime.stop();
    if (err != hipSuccess) { fprintf(stderr, "Error in hipMemcpy device to host output: %s\n", hipGetErrorString( err )); }
 
    err = hipFree(d_input);
    if (err != hipSuccess) { fprintf(stderr, "Error in freeing d_input: %s\n", hipGetErrorString( err )); }
    err = hipFree(d_output);
    if (err != hipSuccess) { fprintf(stderr, "Error in freeing d_output: %s\n", hipGetErrorString( err )); }
    err = hipFree(d_filter);
    if (err != hipSuccess) { fprintf(stderr, "Error in freeing d_filter: %s\n", hipGetErrorString( err )); }

    cout << "convolution (kernel): \t\t" << kernelTime << endl;
    cout << "convolution (memory): \t\t" << memoryTime << endl;

}

int compare_arrays(float *a1, float *a2, int n) {
    int errors = 0;
    int print = 0;

    for (int i=0; i<n; i++) {

        if (isnan(a1[i]) || isnan(a2[i])) {
            errors++;
            if (print < 10) {
                print++;
                fprintf(stderr, "Error NaN detected at i=%d,\t a1= %10.7e \t a2= \t %10.7e\n",i,a1[i],a2[i]);
            }
        }

        float diff = (a1[i]-a2[i])/a1[i];
        if (diff > 1e-6f) {
            errors++;
            if (print < 10) {
                print++;
                fprintf(stderr, "Error detected at i=%d, \t a1= \t %10.7e \t a2= \t %10.7e \t rel_error=\t %10.7e\n",i,a1[i],a2[i],diff);
            }
        }

    }

    return errors;
}

void die(const char *msg){
    if (errno != 0) 
        perror(msg);
    else
        fprintf(stderr, "error: %s\n", msg);
    exit(1);
}

static void readpgm_float(const char *fname,
                          long int height, long int width, float *input) {
    char format[3];
    FILE *f;
    unsigned imgw, imgh, maxv, v;
    size_t i;

    if (!(f = fopen(fname, "r"))) die("fopen");

    fscanf(f, "%2s", format);
    if (format[0] != 'P' || format[1] != '2') die("only ASCII PGM input is supported");

    if (fscanf(f, "%u", &imgw) != 1 ||
        fscanf(f, "%u", &imgh) != 1 ||
        fscanf(f, "%u", &maxv) != 1) die("invalid input");

    if (imgw != width || imgh != height) {
        fprintf(stderr, "input data size (%ux%u) does not match cylinder size (%zux%zu)\n",
                imgw, imgh, width, height);
        die("invalid input");
    }

    for (i = 0; i < width * height; ++i)
    {
        if (fscanf(f, "%u", &v) != 1) die("invalid data");
        input[i] = 0.0 + (float)v * 1.0 / maxv;
    }

    fclose(f);
}

int main(int argc, char *argv[]) {
    int c;
    int i; 
    int errors=0;

    const char *image_path = 0;
    image_path ="../../images/pat1_100x150.pgm";
    int gen_image = 0;

    long int image_height = 1024;
    long int image_width = 1024;

    /* Read command-line options. */
    while((c = getopt(argc, argv, "i:r:h:w")) != -1) {
        switch(c) {
            case 'i':
                image_path = optarg;
            	break;
            case 'r':
            	gen_image = 1;
            	break;
            case 'h':
                image_height = strtol(optarg, 0, 10);
            	break;
            case 'w':
                image_width = strtol(optarg, 0, 10);
				break;
            case '?':
                fprintf(stderr, "Unknown option character '\\x%x'.\n", optopt);
                return -1;
            default:
                return -1;
        }
    }

    long int input_height = (image_height + border_height);
    long int input_width  = (image_width  + border_width);

    //allocate arrays and fill them
    float *input = (float *) malloc(input_height * input_width * sizeof(float));
    float *output1 = (float *) calloc(image_height * image_width, sizeof(float));
    float *output2 = (float *) calloc(image_height * image_width, sizeof(float));
    float *filter = (float *) malloc(filter_height * filter_width * sizeof(float));

    if (gen_image) {
        for (i=0; i< input_height * input_width; i++) {
            input[i] = (float) (i % SEED);
        }
    } else {
        readpgm_float(image_path, image_height, image_width, input);
    }

    //This is specific for a W==H smoothing filter, where W and H are odd.
    for (i=0; i<filter_height * filter_width; i++) { 
      filter[i] = 1.0;
    }

    for (i=filter_width+1; i<(filter_height - 1) * filter_width; i++) {
	    if (i % filter_width > 0 && i % filter_width < filter_width-1)
            filter[i]+=1.0; 
    }

    filter[filter_width*filter_height/2]=3.0;
    //end initialization
   
    //measure the CPU function
    convolutionSeq(output1, input, filter, image_height, image_width, input_height, input_width);

    //measure the GPU function
    convolutionCUDA(output2, input, filter, image_height, image_width, input_height, input_width);


    //check the result
    errors += compare_arrays(output1, output2, image_height*image_width);
    if (errors > 0) {
        printf("TEST FAILED! %d errors!\n", errors);
    } else {
        printf("TEST PASSED!\n");
    }

    free(filter);
    free(input);
    free(output1);
    free(output2);

    return 0;
}


