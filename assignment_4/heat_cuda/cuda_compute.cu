#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <hip/hip_runtime.h>
extern "C" 
{
    #include "compute.h"
}

#define BLOCK_SIZE 512

__device__ __constant__ double COEF_D = 0.1035533905932;
__device__ __constant__ double COEF_S = 0.1464466094067;

/**
 * @brief Copy side columns in the temperature matrix.
 * This function is supposed to be called with a thread per each row.
 * 
 * @param m_heat Pointer to the temperature matrix.
 * @param n_cols Number of columns in the matrix.
 * @param n_rows Number of rows in the matrix.
 * @return __global__ 
 */
__global__ void k_copy_columns(double* m_heat, size_t n_cols, size_t n_rows)
{
    size_t g_idx = blockIdx.x * blockDim.x + threadIdx.x;
 
    if (g_idx >= n_rows) return;
  
    size_t row_idx = g_idx * n_cols;

    m_heat[row_idx]              = m_heat[row_idx + n_cols - 2];
    m_heat[row_idx + n_cols - 1] = m_heat[row_idx + 1];
}

__global__ void k_compute_temp(
    double* m_heat_prev, 
    double* m_heat_next, 
    double* m_coef,
    size_t n_cols, size_t n_cols_actual,
    size_t n_rows, size_t n_rows_actual)
{
    size_t idx_cell = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx_cell < n_rows * n_cols) 
    {
        size_t row = idx_cell / n_cols;
        size_t col = idx_cell - (n_cols * row);
        
        size_t idx_cell_actual          = ((row + 1) * n_cols_actual) + col + 1;
        size_t idx_cell_actual_prev_row = idx_cell_actual - n_cols_actual;
        size_t idx_cell_actual_next_row = idx_cell_actual + n_cols_actual;

        double coef      = m_coef[idx_cell_actual];
        double prev_heat = m_heat_prev[idx_cell_actual];            

        double sum_s = m_heat_prev[idx_cell_actual_prev_row]
                    + m_heat_prev[idx_cell_actual + 1]
                    + m_heat_prev[idx_cell_actual_next_row]
                    + m_heat_prev[idx_cell_actual - 1];
        double sum_d = m_heat_prev[idx_cell_actual_prev_row + 1]
                    + m_heat_prev[idx_cell_actual_next_row + 1]
                    + m_heat_prev[idx_cell_actual_next_row - 1]
                    + m_heat_prev[idx_cell_actual_prev_row - 1];

        double next_heat = (1.0 - coef) * (sum_d * COEF_D + sum_s * COEF_S) + coef * prev_heat;

        m_heat_next[idx_cell_actual] = next_heat;
    }
}

extern "C" 
void cuda_do_compute(const struct parameters *p, struct results *r)
{
    struct timespec before, after;

    int i = 1;

    int n_cols       = p->M;
    int n_rows       = p->N;
    int n_cells      = n_rows * n_cols;
    int n_iters      = p->maxiter;
    int n_report     = p->period;
    int printreports = p->printreports;

    // we initialize this to enter the while loop
    r->maxdiff = p->threshold;

    int n_cols_actual  = n_cols + 2;
    int n_rows_actual  = n_rows + 2;
    int n_cells_actual = n_cols_actual * n_rows_actual;

    double *m_heat_prev = (double*) malloc(n_cells_actual * sizeof(double));
    double *m_heat_next = (double*) malloc(n_cells_actual * sizeof(double));
    double *m_coef   = (double*) malloc(n_cells_actual * sizeof(double));

    double *m_heat_prev_device = NULL;
    double *m_heat_next_device = NULL;
    double *m_coef_device      = NULL;

    if (hipMalloc(&m_heat_prev_device, n_cells_actual * sizeof(double)) != hipSuccess) goto end;
    if (hipMalloc(&m_heat_next_device, n_cells_actual * sizeof(double)) != hipSuccess) goto end;
    if (hipMalloc(&m_coef_device,      n_cells_actual * sizeof(double)) != hipSuccess) goto end;

    if (m_heat_prev_device == NULL || m_heat_next_device == NULL || m_coef_device == NULL) goto end;

    // copy real matrix
    for (int row = 0; row < n_rows; ++row)
    {
        for (int col = 0; col < n_cols; ++col)
        {
            m_heat_prev[(row + 1) * n_cols_actual + (col + 1)] = p->tinit[row * n_cols + col];
            m_coef[(row + 1) * n_cols_actual + (col + 1)] = p->conductivity[row * n_cols + col];
        }
    }

    // copy halo
    for (int col = 0; col < n_cols_actual; ++col)
    {
        m_heat_prev[col] = m_heat_prev[n_cols_actual + col];
        m_heat_prev[(n_rows_actual - 1) * n_cols_actual + col] = m_heat_prev[(n_rows_actual - 2) * n_cols_actual + col]; 
    }

    memcpy(m_heat_next, m_heat_prev, n_cells_actual * sizeof(double));

    if (hipMemcpy(m_heat_prev_device, m_heat_prev, n_cells_actual * sizeof(double), hipMemcpyHostToDevice) != hipSuccess) goto end;
    if (hipMemcpy(m_heat_next_device, m_heat_next, n_cells_actual * sizeof(double), hipMemcpyHostToDevice) != hipSuccess) goto end;    
    if (hipMemcpy(m_coef_device,      m_coef,      n_cells_actual * sizeof(double), hipMemcpyHostToDevice) != hipSuccess) goto end;

    clock_gettime(CLOCK_MONOTONIC, &before);

    while (i < n_iters + 1 && r->maxdiff >= p-> threshold)
    {
        // swap buffers
        { 
            double *temp = m_heat_prev_device; 
            m_heat_prev_device = m_heat_next_device; 
            m_heat_next_device = temp; 
        }

        // copy border cells
        k_copy_columns<<<n_rows_actual / BLOCK_SIZE + 1, BLOCK_SIZE>>>(m_heat_prev_device, n_cols_actual, n_rows_actual);
        // perform simulation
        k_compute_temp<<<n_cells / BLOCK_SIZE + 1, BLOCK_SIZE>>>(
            m_heat_prev_device, 
            m_heat_next_device, 
            m_coef_device, 
            n_cols, n_cols_actual, 
            n_rows, n_rows_actual);

        if (i % n_report == 0 || i == n_iters) 
        {
            hipDeviceSynchronize();

            clock_gettime(CLOCK_MONOTONIC, &after);

            if (hipMemcpy(m_heat_prev, m_heat_prev_device, n_cells_actual * sizeof(double), hipMemcpyDeviceToHost) != hipSuccess) goto end;
            if (hipMemcpy(m_heat_next, m_heat_next_device, n_cells_actual * sizeof(double), hipMemcpyDeviceToHost) != hipSuccess) goto end;    

            double heat_sum = 0.0;

            r->niter    = i;
            r->tmax     = -INFINITY;
            r->tmin     = INFINITY;
            r->maxdiff  = 0.0;

            for (int row = 1; row < n_rows_actual - 1; ++row)
            {
                int idx_row = row * n_cols_actual;

                for (int col = 1; col < n_cols_actual - 1; ++col)
                {   
                    double next_heat = m_heat_next[idx_row + col];
                    double prev_heat = m_heat_prev[idx_row + col];

                    double heat_abs_diff = fabs(prev_heat - next_heat);

                    r->tmax     = fmax(r->tmax, next_heat);
                    r->tmin     = fmin(r->tmin, next_heat);
                    r->maxdiff  = fmax(r->maxdiff, heat_abs_diff);

                    heat_sum += next_heat;
                }
            }

            r->tavg  = heat_sum / (double) n_cells;
            r->time  = (double)(after.tv_sec - before.tv_sec) +
                       (double)(after.tv_nsec - before.tv_nsec) / 1e9;

            if (printreports && i != n_iters) report_results(p, r);
        }

        i++;
    }

end:

    hipFree(m_coef_device);
    hipFree(m_heat_prev_device);
    hipFree(m_heat_next_device);

    free(m_coef);
    free(m_heat_prev);
    free(m_heat_next);
}
