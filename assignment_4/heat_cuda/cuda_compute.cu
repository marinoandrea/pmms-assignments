#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <hip/hip_runtime.h>
extern "C" 
{
    #include "compute.h"
}

#define BLOCK_SIZE 128
#define BLOCK_SIZE_REPORT 128

__device__ __constant__ double COEF_D = 0.1035533905932;
__device__ __constant__ double COEF_S = 0.1464466094067;

/**
 * @brief Copy side columns in the temperature matrix.
 * This function is supposed to be called with a thread per each row.
 * 
 * @param m_heat Pointer to the temperature matrix.
 * @param n_cols Number of columns in the matrix.
 * @param n_rows Number of rows in the matrix.
 * @return __global__ 
 */
__global__ void k_copy_columns(double* m_heat, size_t n_cols, size_t n_rows)
{
    size_t g_idx = blockIdx.x * blockDim.x + threadIdx.x;
 
    if (g_idx < n_rows)
    {
        size_t row_idx = g_idx * n_cols;

        m_heat[row_idx]              = m_heat[row_idx + n_cols - 2];
        m_heat[row_idx + n_cols - 1] = m_heat[row_idx + 1];
    }
}

__global__ void k_compute_temp(
    double* m_heat_prev, 
    double* m_heat_next, 
    double* m_coef,
    size_t n_cols, size_t n_cols_actual,
    size_t n_rows, size_t n_rows_actual)
{
    size_t idx_cell = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx_cell < n_rows * n_cols) 
    {
        size_t row = idx_cell / n_cols;
        size_t col = idx_cell - (n_cols * row);
        
        size_t idx_cell_actual          = ((row + 1) * n_cols_actual) + col + 1;
        size_t idx_cell_actual_prev_row = idx_cell_actual - n_cols_actual;
        size_t idx_cell_actual_next_row = idx_cell_actual + n_cols_actual;

        double coef      = m_coef[idx_cell_actual];
        double prev_heat = m_heat_prev[idx_cell_actual];            

        double sum_s = m_heat_prev[idx_cell_actual_prev_row]
                    + m_heat_prev[idx_cell_actual + 1]
                    + m_heat_prev[idx_cell_actual_next_row]
                    + m_heat_prev[idx_cell_actual - 1];
        double sum_d = m_heat_prev[idx_cell_actual_prev_row + 1]
                    + m_heat_prev[idx_cell_actual_next_row + 1]
                    + m_heat_prev[idx_cell_actual_next_row - 1]
                    + m_heat_prev[idx_cell_actual_prev_row - 1];

        double next_heat = (1.0 - coef) * (sum_d * COEF_D + sum_s * COEF_S) + coef * prev_heat;

        m_heat_next[idx_cell_actual] = next_heat;
    }
}

__global__ void k_compute_report(
    double* m_heat_prev, 
    double* m_heat_next, 
    size_t n_cols, size_t n_cols_actual,
    size_t n_rows, size_t n_rows_actual,
    double *out_g_sums,
    double *out_g_difs,
    double *out_g_mins,
    double *out_g_maxs)
{
    __shared__ double s_data[BLOCK_SIZE_REPORT * sizeof(double) * 4];

    double *s_sums = s_data;
    double *s_difs = &s_data[BLOCK_SIZE];
    double *s_mins = &s_data[BLOCK_SIZE * 2];
    double *s_maxs = &s_data[BLOCK_SIZE * 3];

    unsigned int tid = threadIdx.x;
    unsigned int gid = blockIdx.x * blockDim.x + threadIdx.x;

    s_sums[tid] = 0.0;
    s_mins[tid] = INFINITY;
    s_maxs[tid] = -INFINITY;
    s_difs[tid] = -INFINITY;

    if (gid < n_rows * n_cols)
    {
        size_t row = gid / n_cols;
        size_t col = gid - (n_cols * row);
        size_t idx_cell_actual = ((row + 1) * n_cols_actual) + col + 1;

        s_sums[tid] = m_heat_next[idx_cell_actual];
        s_mins[tid] = m_heat_next[idx_cell_actual];
        s_maxs[tid] = m_heat_next[idx_cell_actual];
        s_difs[tid] = fabs(m_heat_next[idx_cell_actual] - m_heat_prev[idx_cell_actual]);
        __syncthreads();

        for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) 
        {
            if (tid < s)
            {
                s_sums[tid] += s_sums[tid + s];
                s_mins[tid] = fmin(s_mins[tid + s], s_mins[tid]);
                s_maxs[tid] = fmax(s_maxs[tid + s], s_maxs[tid]);
                s_difs[tid] = fmax(s_difs[tid + s], s_difs[tid]);
            }
            __syncthreads();
        }

        if (tid == 0)
        {
            out_g_sums[blockIdx.x] = s_sums[0];
            out_g_mins[blockIdx.x] = s_mins[0];
            out_g_maxs[blockIdx.x] = s_maxs[0];
            out_g_difs[blockIdx.x] = s_difs[0];
        }
    }
}


extern "C" 
void cuda_do_compute(const struct parameters *p, struct results *r)
{
    struct timespec before, after;

    int i = 1;

    int n_cols       = p->M;
    int n_rows       = p->N;
    int n_cells      = n_rows * n_cols;
    int n_iters      = p->maxiter;
    int n_report     = p->period;
    int printreports = p->printreports;

    unsigned int n_blocks = n_cells / BLOCK_SIZE + 1;

    // we initialize this to enter the while loop
    r->maxdiff = p->threshold;

    int n_cols_actual  = n_cols + 2;
    int n_rows_actual  = n_rows + 2;
    int n_cells_actual = n_cols_actual * n_rows_actual;

    // host temperature and coefficient matrices
    double *m_heat_prev = (double*) malloc(n_cells_actual * sizeof(double));
    double *m_heat_next = (double*) malloc(n_cells_actual * sizeof(double));
    double *m_coef      = (double*) malloc(n_cells_actual * sizeof(double));

    // host redution buffers
    double *sums = (double*) malloc(n_blocks * sizeof(double));
    double *difs = (double*) malloc(n_blocks * sizeof(double));
    double *mins = (double*) malloc(n_blocks * sizeof(double));
    double *maxs = (double*) malloc(n_blocks * sizeof(double));

    // device temperature and coefficient matrices
    double *m_heat_prev_device = NULL;
    double *m_heat_next_device = NULL;
    double *m_coef_device      = NULL;

    // device redution buffers
    double *sums_device = NULL;
    double *difs_device = NULL;
    double *mins_device = NULL;
    double *maxs_device = NULL;

    // allocate device global memory temperature and coefficient matrices
    if (hipMalloc(&m_heat_prev_device, n_cells_actual * sizeof(double)) != hipSuccess) goto end;
    if (hipMalloc(&m_heat_next_device, n_cells_actual * sizeof(double)) != hipSuccess) goto end;
    if (hipMalloc(&m_coef_device,      n_cells_actual * sizeof(double)) != hipSuccess) goto end;

    if (m_heat_prev_device == NULL || m_heat_next_device == NULL || m_coef_device == NULL) goto end;

    // allocate device global memory reduction buffers
    if (hipMalloc(&sums_device, n_blocks * sizeof(double)) != hipSuccess) goto end;
    if (hipMalloc(&difs_device, n_blocks * sizeof(double)) != hipSuccess) goto end;
    if (hipMalloc(&mins_device, n_blocks * sizeof(double)) != hipSuccess) goto end;
    if (hipMalloc(&maxs_device, n_blocks * sizeof(double)) != hipSuccess) goto end;

    if (sums_device == NULL || difs_device == NULL || mins_device == NULL || maxs_device == NULL) goto end;

    // copy real matrix
    for (int row = 0; row < n_rows; ++row)
    {
        for (int col = 0; col < n_cols; ++col)
        {
            m_heat_prev[(row + 1) * n_cols_actual + (col + 1)] = p->tinit[row * n_cols + col];
            m_coef[(row + 1) * n_cols_actual + (col + 1)] = p->conductivity[row * n_cols + col];
        }
    }

    // copy halo
    for (int col = 0; col < n_cols_actual; ++col)
    {
        m_heat_prev[col] = m_heat_prev[n_cols_actual + col];
        m_heat_prev[(n_rows_actual - 1) * n_cols_actual + col] = m_heat_prev[(n_rows_actual - 2) * n_cols_actual + col]; 
    }

    memcpy(m_heat_next, m_heat_prev, n_cells_actual * sizeof(double));

    if (hipMemcpy(m_heat_prev_device, m_heat_prev, n_cells_actual * sizeof(double), hipMemcpyHostToDevice) != hipSuccess) goto end;
    if (hipMemcpy(m_heat_next_device, m_heat_next, n_cells_actual * sizeof(double), hipMemcpyHostToDevice) != hipSuccess) goto end;    
    if (hipMemcpy(m_coef_device,      m_coef,      n_cells_actual * sizeof(double), hipMemcpyHostToDevice) != hipSuccess) goto end;

    clock_gettime(CLOCK_MONOTONIC, &before);

    while (i < n_iters + 1 && r->maxdiff >= p-> threshold)
    {
        // swap buffers
        { 
            double *temp = m_heat_prev_device; 
            m_heat_prev_device = m_heat_next_device; 
            m_heat_next_device = temp; 
        }

        // copy border cells
        k_copy_columns<<<n_rows_actual / BLOCK_SIZE + 1, BLOCK_SIZE>>>(m_heat_prev_device, n_cols_actual, n_rows_actual);
        // perform simulation
        k_compute_temp<<<n_blocks, BLOCK_SIZE>>>(
            m_heat_prev_device, 
            m_heat_next_device, 
            m_coef_device, 
            n_cols, n_cols_actual, 
            n_rows, n_rows_actual);
            
        if (i % n_report == 0 || i == n_iters) 
        {
            hipDeviceSynchronize();

            clock_gettime(CLOCK_MONOTONIC, &after);

            k_compute_report<<<n_cells / BLOCK_SIZE_REPORT + 1, BLOCK_SIZE_REPORT>>>(
                m_heat_prev_device, 
                m_heat_next_device, 
                n_cols, n_cols_actual, 
                n_rows, n_rows_actual,
                sums_device,
                difs_device,
                mins_device,
                maxs_device);

            if (hipMemcpy(sums, sums_device, n_blocks * sizeof(double), hipMemcpyDeviceToHost) != hipSuccess) goto end;
            if (hipMemcpy(mins, mins_device, n_blocks * sizeof(double), hipMemcpyDeviceToHost) != hipSuccess) goto end;    
            if (hipMemcpy(maxs, maxs_device, n_blocks * sizeof(double), hipMemcpyDeviceToHost) != hipSuccess) goto end;
            if (hipMemcpy(difs, difs_device, n_blocks * sizeof(double), hipMemcpyDeviceToHost) != hipSuccess) goto end;

            // complete reduction on CPU
            double heat_sum = 0.0;

            r->niter    = i;
            r->tmax     = -INFINITY;
            r->tmin     = INFINITY;
            r->maxdiff  = -INFINITY;

            for (int idx = 0; idx < n_blocks; ++idx)
            {                
                r->tmax     = fmax(r->tmax,    maxs[idx]);
                r->tmin     = fmin(r->tmin,    mins[idx]);
                r->maxdiff  = fmax(r->maxdiff, difs[idx]);
                heat_sum    += sums[idx];
            }

            r->tavg  = heat_sum / (double) n_cells;
            r->time  = (double)(after.tv_sec - before.tv_sec) +
                       (double)(after.tv_nsec - before.tv_nsec) / 1e9;

            if (printreports && i != n_iters) report_results(p, r);
        }

        i++;
    }

end:

    hipFree(sums_device);
    hipFree(mins_device);
    hipFree(maxs_device);
    hipFree(difs_device);

    hipFree(m_coef_device);
    hipFree(m_heat_prev_device);
    hipFree(m_heat_next_device);

    free(sums);
    free(mins);
    free(maxs);
    free(difs);

    free(m_coef);
    free(m_heat_prev);
    free(m_heat_next);
}
